
#include <hip/hip_runtime.h>
//#define _CRT_SECURE_NO_WARNINGS
//
//#include"imgProcess.h"
////#include "imgProcess.cu"
//#include"kexinLibs.h"
//
//#include<vector>
//#include<iostream>
//
//#include"Timer.h"
//
//using std::string;
//using std::vector;
//using std::cout;
//using std::endl;
//
//
//
//int main()
//{
//	//计时
//	Timer timer1;
//	
//
//
//	//读取PSF和未重构的文件
//	string PSF_1_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/PSF_1_zhuanzhi_float.dat";//matlab中保存出来的float类型
//	string X31_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/r20210924_2_X31_resize.tif";
//	//读取角度、二维模板信息
//	string rotationAngleXY_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/rotationAngleXY.dat";//360个double
//	string rotationAngleYZ_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/rotationAngleYZ.dat";//31个double
//	string template_roXY_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/templateXY.tif";//200*200*360个float，按照matlab中行优先存储，存完一个波段再存第二个波段
//	string template_roYZ_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/template_roYZ.dat";//200*50*31个float，按照matlab中行优先存储，存完一个波段再存第二个波段
//	//读取用于仿射对齐的fixImage
//	string fixImage_file = "D:/kexin/Online-Zebrafish-Optogenetic/data/old/toAffineWithZBB.tif";
//
//
//	string imgBeforeRecon_path = "D:/kexin/Online-Zebrafish-Optogenetic/data/r20210824_X10_4bin";
//	vector<string> imgBeforeRecon_Names;
//	getFileNames(imgBeforeRecon_path, imgBeforeRecon_Names);
//
//	string modelPath = "affineNetScript_TM_0621_3080.pt";
//
//	FishImageProcess fishImgProc(modelPath);
//	
//	fishImgProc.readPSFfromFile(PSF_1_file);
//	fishImgProc.readRotationAngleFromFile(rotationAngleXY_file, rotationAngleYZ_file);
//	fishImgProc.readTemplateFromFile(template_roXY_file, template_roYZ_file);
//	fishImgProc.readFixImageFromFile(fixImage_file);
//	                                                                                                                                   
//	fishImgProc.initializeFishReg("anatomyList_4bin.txt");
//
//	fishImgProc.prepareGPUMemory();
//	fishImgProc.processPSF();
//
//	for (int i = 0; i < imgBeforeRecon_Names.size(); i++)
//	{
//		fishImgProc.readImageFromFile(imgBeforeRecon_Names[i]);
//
//		timer1.start();
//		fishImgProc.reconImage();//重构读进来的图像
//		fishImgProc.cropReconImage();   
//
//		//rotation
//		fishImgProc.matchingANDrotationXY();
//
//		//crop
//		fishImgProc.cropRotatedImage();
//
//		////crop的结果构建movingTensor，和fixTensor一起送入网络处理
//		fishImgProc.libtorchModelProcess();
//
//		////结合rotation/crop/affine的数据做坐标转换
//		std::vector<cv::Point3f> points = fishImgProc.ZBB2FishTransform();
//
//
//		timer1.stop();
//		cout << "time cost: " << timer1.getElapsedTimeInMilliSec() << " ms" << endl;
//
//		//for (int j = 0; j < points.size(); j++)
//		//{
//		//	cout << points[j] << endl;
//		//}
//
//		//save and check
//		string saveName1 = "D:/kexin/Online-Zebrafish-Optogenetic/data/testRecon/" + int2string(4, i) + ".tif";
//		string saveName2 = "D:/kexin/Online-Zebrafish-Optogenetic/data/testMatchingXY/" + int2string(4, i) + ".tif";
//		string saveName3 = "D:/kexin/Online-Zebrafish-Optogenetic/data/testCrop/" + int2string(4, i) + ".tif";
//		
//		//76 * 95 * 50
//		//200*200*50
//		//test reconstruction
//		float* temp = new float[200 * 200 * 50]();
//		cudaMemcpy(temp, fishImgProc.gpuObjRecon_crop, sizeof(float) * 200 * 200 * 50, cudaMemcpyDeviceToHost);
//		saveAndCheckImage(temp, 200, 200, 50, saveName1);
//		//test rotation
//		float* temp1 = new float[200 * 200 * 50]();
//		cudaMemcpy(temp1, fishImgProc.imageRotated3D_gpu, sizeof(float) * 200 * 200 * 50, cudaMemcpyDeviceToHost);
//		saveAndCheckImage(temp1, 200 , 200 , 50, saveName2);
//		//test crop
//		float* temp2 = new float[76 * 95 * 50]();
//		cudaMemcpy(temp2, fishImgProc.ObjCropRed_gpu, sizeof(float) * 76 * 95 * 50, cudaMemcpyDeviceToHost);
//		saveAndCheckImage(temp2, 76 , 95 , 49, saveName3);
//
//		free(temp);
//		free(temp1);
//		free(temp2);
//	}
//
//	fishImgProc.freeMemory();
//
//	return 0;
//}